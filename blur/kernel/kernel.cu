#include <stdio.h>
#include "kernel.hh"
#include "gaussian.cu"

template <typename T>
void GaussianBlurCaller(const T * img, T * dst, const int kw, const int center, float * kernel)
{
	T * d_img;
	T * d_dst;
	float * d_kernel;
	hipMalloc(&d_img, sizeof(T));
	hipMalloc(&d_dst, sizeof(T));
	hipMalloc(&d_kernel, kw*sizeof(float));
	hipMemcpy(d_img, img, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, kw*sizeof(float), hipMemcpyHostToDevice);

	GaussianBlurKernel<<<1, 1>>>(d_img, d_dst, kw, center, d_kernel);

	hipFree(d_img);
	hipFree(d_dst);
	hipFree(d_kernel);
}
