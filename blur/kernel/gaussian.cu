
#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
__global__ void GaussianBlurKernel(const T * img, T * dst, const int kw, const int center, float * kernel)
{
	printf("%d\f", center);
}


template <typename T>
void GaussianBlurInit(const T * img, T * dst, const int kw, const int center, float * kernel)
{
	T * d_img;
	T * d_dst;
	float * d_kernel;
	hipMalloc(&d_img, sizeof(T));
	hipMalloc(&d_dst, sizeof(T));
	hipMalloc(&d_kernel, kw*sizeof(float));
	hipMemcpy(d_img, img, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, kw*sizeof(float), hipMemcpyHostToDevice);

	GaussianBlurKernel<<<1, 1>>>(d_img, d_dst, kw, center, d_kernel);

	hipFree(d_img);
	hipFree(d_dst);
	hipFree(d_kernel);
}
