#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "gaussian.cuh"

__global__ void GaussianBlurKernel(const float * img, float * dst, const int kw, const int center, float * kernel)
{
	printf("%d\n", center);
}

void GaussianBlurCaller(const float * img, float * dst, const int kw, const int center, float * kernel) {
	float * d_img;
	float * d_dst;
	float * d_kernel;
	hipMalloc(&d_img, sizeof(float));
	hipMalloc(&d_dst, sizeof(float));
	hipMalloc(&d_kernel, sizeof(float)*kw);
	hipMemcpy(d_img, img, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, sizeof(float)*kw, hipMemcpyHostToDevice);

	GaussianBlurKernel<<<1, 1>>>(d_img, d_dst, kw, center, d_kernel);

	hipFree(d_img);
	hipFree(d_dst);
	hipFree(d_kernel);
}
