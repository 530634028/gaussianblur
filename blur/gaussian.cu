
#include <hip/hip_runtime.h>
#define tx threadIdx.x
#define ty threadIdx.y
#define bx blockIdx.x
#define by blockIdx.y
#define bdx blockDim.x
#define bdy blockDim.y

#define MAX_KSIZE_HALF 100

__constant__ float c_gKer[MAX_KSIZE_HALF+1];

/*
template <typename T>
__global__ void gaussian_blur_kernel (const int height,
                                      const int width,
                                      const int center,
                                      const T * src,
                                      const T * dst)
{
	const int y = by * bdy + ty;
	const int x = bx * bdx + tx;
	extern __shared__ float smem[];
	volatile float * row = smem + ty * (bdx + 2 * center);
	
	if (y < height)
	{
		for (int i = tx; i < bdx + 2 * center; i += bdx)
		{
			int x_ext = int(bx * bdx) + i - center;
			x_ext = b.idx_col(x_ext);
			row[i] = src(y, x_ext) * c_gKer[0];
			for (int j = 1; j <= center; ++j)
			{
				row[i] += (src(b.idx_row_low(y-j), x_ext) +
                                           src(b.idx_row_high(y+j), x_ext)) * c_gKer[j];
			}
		}

		if (x < width)
		{
			__syncthreads();
			// horizontal pass
			row += tx + center;
			float res = row[0] * c_gKer[0];
			for (int i = 1; i <= center; ++i)
			{
				res += (row[-i] + row[i]) * c_gKer[i];
			}
			dst(y, x) = res;
		}
	} 
}
*/


/* --------------------------------------------------------------------------- */

template <typename T>
__global__ void GaussianBlurKernel (const int center, const int w, const int h)
{
	const int memsize = center * 2 + max(w, h);
	float * cur_line_mem[memsize];
	hipMalloc(cur_line_mem, memsize*sizeof(T));
	volatile T *cur_line = cur_line_mem + ty * (bdx + 2 * center);
}
