#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


#define max(a, b) ((a) > (b) ? a : b)
#define min(a, b) ((a) < (b) ? a : b)


__global__ void GaussianBlurKernel(const float * img, float * dst, const int width, const int height,
                                   const int kw, const int center, float * kernel) 
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (width*height); i += blockDim.x * gridDim.x)
	{
		const int pw = i % kw;
		const int ph = (i / kw) % kw;
		const int n = i / kw / kw;
		int hstart = ph - center;
		int wstart = pw - center;
		int hend = min(hstart + kw, height + center);
		int wend = min(wstart + kw, width + center);
		hstart = max(hstart, 0);
		wstart = max(wstart, 0);
		hend = min(hend, height);
		wend = min(wend, width);

		float val = 0;
		const float * in_slice = img + n * height * width;
		int counter = 0;
		for (int h = hstart; h < hend; ++h)
		{
			for (int w = wstart; w < wend; ++w)
			{
				val += in_slice[h * width + w] * kernel[counter];
				counter++;
			}
		}
		dst[i] = val;
	}
}


void GaussianBlurCaller(const float * img, float * dst, const int w, const int h,
                        const int kw, const int center, float * kernel) 
{
	float * d_img;
	float * d_dst;
	float * d_kernel;
	hipMalloc(&d_img, sizeof(float)*w*h);
	hipMalloc(&d_dst, sizeof(float)*w*h);
	hipMalloc(&d_kernel, sizeof(float)*kw*kw);
	hipMemcpy(d_img, img, sizeof(float)*w*h, hipMemcpyHostToDevice);
	hipMemcpy(d_dst, dst, sizeof(float)*w*h, hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, sizeof(float)*kw*kw, hipMemcpyHostToDevice);

	int threadsPerBlock = 512;
	int numBlocks = (w * h + threadsPerBlock - 1) / threadsPerBlock;
	GaussianBlurKernel<<<numBlocks, threadsPerBlock>>>(d_img, d_dst, w, h, kw, center, d_kernel);
	hipMemcpy(dst, d_dst, sizeof(float)*w*h, hipMemcpyDeviceToHost);

	hipFree(d_img);
	hipFree(d_dst);
	hipFree(d_kernel);
}
